#include "hip/hip_runtime.h"
#include <unordered_map>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <algorithm>

typedef unsigned long long uint64;
typedef unsigned int uint32;
typedef unsigned char uint8;

using namespace std;

#define mix(a,b,c) \
{ \
  a -= b; a -= c; a ^= (c>>13); \
  b -= c; b -= a; b ^= (a<<8); \
  c -= a; c -= b; c ^= (b>>13); \
  a -= b; a -= c; a ^= (c>>12);  \
  b -= c; b -= a; b ^= (a<<16); \
  c -= a; c -= b; c ^= (b>>5); \
  a -= b; a -= c; a ^= (c>>3);  \
  b -= c; b -= a; b ^= (a<<10); \
  c -= a; c -= b; c ^= (b>>15); \
}

#define MIN(a, b) ((a)<(b)?(a):(b))

#define OFFSET(a, b, c) ((a)*LEN1*3+(b)*3+(c)) 

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
      fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
      assert(result == hipSuccess);
    }
#endif
    return result;
}

__host__ __device__
uint32 hash1(uint32 key) {
    return (key * 2654435761u) >> 15;
}

__host__ __device__
uint32 hash2(uint32 key) {
        //register ub4 a,b,c,len;
    uint32 a,b,c;
    uint32 len = 4;
    char* str = (char*)&key;
    //  uint32_t initval = 0;
    /* Set up the internal state */
    //len = length;
    a = b = 0x9e3779b9;  /* the golden ratio; an arbitrary value */
    c = 8311;//prime32[1000];         /* the previous hash value */

    /*---------------------------------------- handle most of the key */
    while (len >= 12)
    {
        a += (str[0] +((uint32)str[1]<<8) +((uint32)str[2]<<16) +((uint32)str[3]<<24));
        b += (str[4] +((uint32)str[5]<<8) +((uint32)str[6]<<16) +((uint32)str[7]<<24));
        c += (str[8] +((uint32)str[9]<<8) +((uint32)str[10]<<16)+((uint32)str[11]<<24));
        mix(a,b,c);
        str += 12; len -= 12;
    }

    /*------------------------------------- handle the last 11 bytes */
    c += len;
    switch(len)              /* all the case statements fall through */
    {
        case 11: c+=((uint32)str[10]<<24);
        // fall through
        case 10: c+=((uint32)str[9]<<16);
        // fall through
        case 9 : c+=((uint32)str[8]<<8);
        /* the first byte of c is reserved for the length */
        // fall through
        case 8 : b+=((uint32)str[7]<<24);
        // fall through
        case 7 : b+=((uint32)str[6]<<16);
        // fall through
        case 6 : b+=((uint32)str[5]<<8);
        // fall through
        case 5 : b+=str[4];
        // fall through
        case 4 : a+=((uint32)str[3]<<24);
        // fall through
        case 3 : a+=((uint32)str[2]<<16);
        // fall through
        case 2 : a+=((uint32)str[1]<<8);
        // fall through
        case 1 : a+=str[0];
        /* case 0: nothing left to add */
    }
    mix(a,b,c);
    /*-------------------------------------------- report the result */
    return c;
}

const int LEN1 = 3200;
const int LEN2 = 460800;
const int BLOCK_SIZE = 32;

__device__ uint32 kernel_heavy[4 * LEN1 * 3];
__device__ uint32 kernel_light[LEN2];
__device__ uint32 mutex[4 * LEN1];

uint32 heavy[4 * LEN1 * 3];
uint32 light[LEN2];

uint32* read_keys(char* filename, uint32* len) {
    FILE* fp = fopen(filename, "rb");
    fseek(fp, 0L, SEEK_END);
    uint32 size = ftell(fp);
    rewind(fp);
    if (size % 13) {
        printf("wrong file size: %d\n", size);
        fclose(fp);
        return NULL;
    }
    *len = size / 13;
    uint32* keys;
    checkCuda( hipHostMalloc((void**)&keys, sizeof(uint32) * (*len)) );
    for (int i = 0; i < *len; ++i) {
        char tmp[13];
        fread(tmp, 13, sizeof(char), fp);
        *(keys+i) = *(uint32*)tmp;
    }
    fclose(fp);
    return keys;
}

__global__ void kernel_init() {
    for (int i = 0; i < 4 * LEN1 * 3; ++i)
        kernel_heavy[i] = 0;

    for (int i = 0; i < LEN2; ++i)
        kernel_light[i] = 0;

    for (int i = 0; i < LEN1 * 4; ++i)
        mutex[i] = 0;
}

__device__ inline bool insert_bucket(uint32 offset, uint32* key, uint32* val) {


    uint32* bucket = kernel_heavy + offset;

    bool res;

    bool next = true;
    int cnt = 10;
    while(next && cnt--)
    {

        int v = atomicCAS(mutex + offset / 3, 0, 1);
        if (v == 0)
        {

            if (bucket[0] == 0) {
                bucket[0] = *val;
                bucket[1] = *key;
                res = false;
            }
            else if (bucket[1] == *key) {
                bucket[0] = ((bucket[0] + *val) | (((bucket[0] | *val) >> 31) << 31));
                res = false;
            } else {
                bucket[2] += ((*val<<1)>>1);
                if (bucket[2] < (bucket[0] << 3)) {
                    res = true;
                } else {
                    bucket[2] = 0;
                    uint32 t_key = bucket[1];
                    uint32 t_val = bucket[0];
                    bucket[1] = *key;
                    bucket[0] = ((1u<<31) | *val);
                    *key = t_key;
                    *val = t_val;
                    res = true;
                }
            }

            atomicExch(mutex + offset / 3, 0);
            next = false;
        }
    }
    return res;
}

__device__ bool insert_heavy(uint32* key, uint32* val) {
    uint32 hash_val = hash1(*key);
    for (int i = 0; i < 4; ++i) {
        uint32 pos = hash_val % (LEN1 - i);
        if (insert_bucket(OFFSET(i, pos, 0), key, val) == false)
            return false;
    }
    return true;
}

__device__ inline void insert_light(uint32 key, uint32 val) {
    int pos = hash2(key) % LEN2;
    atomicAdd(kernel_light + pos, val);
}

__global__ void kernel_insert(uint32* keys, uint32 len) {
    const uint32 id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= len) return;
    uint32 key = keys[id];
    uint32 val = 1;
    if (insert_heavy(&key, &val))
        insert_light(key, val);
}

double expr_result[10];
int expr_pos = 0;

void insert(uint32* keys, uint32 len) {
    uint32* kernel_keys;
    /**/checkCuda( hipMalloc((void**)&kernel_keys, sizeof(uint32) * len) );
    //checkCuda( hipMalloc((void**)&kernel_keys, sizeof(uint32) * BATCH_SIZE) );

    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );

    int batch_num = (len + BATCH_SIZE - 1) / BATCH_SIZE;
    hipStream_t stream[batch_num];
    for (int i = 0; i < batch_num; ++i)
        checkCuda( hipStreamCreate(&stream[i]) );


    checkCuda( hipEventRecord(startEvent, 0) );

    for (uint32 i = 0, left = len; i < len; i += BATCH_SIZE, left -= BATCH_SIZE) {
        uint32 size = MIN(left, BATCH_SIZE);
        /**/checkCuda( hipMemcpyAsync(kernel_keys + i, keys + i, sizeof(uint32) * size, hipMemcpyHostToDevice, stream[i % BATCH_SIZE]) );
        /**/kernel_insert<<<(size + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, stream[i % BATCH_SIZE]>>>(kernel_keys + i, size);
        //checkCuda( hipMemcpy(kernel_keys, keys + i, sizeof(uint32) * size, hipMemcpyHostToDevice) );
        //kernel_insert<<<(size + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(kernel_keys, size);
    }

    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    float ms;
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    //printf("%f\n", len / ms / 1000);
    /**/expr_result[expr_pos++] = len / ms / 1000;
    //expr_result[expr_pos++] = ms / batch_num * 1000;
    hipFree(kernel_keys);
}

uint32* true_result(uint32* keys, uint32* len) {
    unordered_map<uint32, uint32> s;
    for (int i = 0; i < *len; ++i) ++s[keys[i]];
    *len = s.size();
    uint32* res = (uint32*)malloc(sizeof(uint32) * (*len));
    int pos = 0;
    for (auto p: s) {
        keys[pos] = p.first;
        res[pos] = p.second;
        ++pos;
    }
    return res;
}

inline bool query_bucket(uint32* bucket, uint32 key, uint32* res) {
    if (bucket[1] != key) return false;
    *res += ((bucket[0] << 1) >> 1);
    return (bucket[0] >> 31) == 1;
}

bool query_heavy(uint32* res, uint32 key) {
    uint32 hash_val = hash1(key);
    *res = 0;
    bool should_query_light = false;
    for (int i = 0; i < 4; ++i) {
        uint32 pos = hash_val % (LEN1 - i);
        bool t = query_bucket(heavy + OFFSET(i, pos, 0), key, res);
        should_query_light = (should_query_light || t);
    }
    return should_query_light;
}

void query_light(uint32* res, uint32 key) {
    uint32 pos = hash2(key) % LEN2;
    *res += (light[pos] & 0xff);
}

uint32* query(uint32* keys, uint32 len) {
    uint32* res = (uint32*)malloc(sizeof(uint32) * len);
    checkCuda( hipMemcpyFromSymbol(heavy, HIP_SYMBOL(kernel_heavy), 4 * LEN1 * 3 * sizeof(uint32)) );
    checkCuda( hipMemcpyFromSymbol(light, HIP_SYMBOL(kernel_light), LEN2 * sizeof(uint32)) );
    for (int i = 0; i < len; ++i)
        if (query_heavy(res + i, keys[i]) || res[i] == 0)
            query_light(res + i, keys[i]);
    return res;
}

void compare(uint32* real, uint32* my, uint32 len) {
    int accuracy = 0;
    double are = 0;
    for (int i = 0; i < len; ++i) {
        accuracy += (real[i] == my[i]);
        are += ((real[i] > my[i]) ? (real[i] - my[i]) : (my[i] - real[i])) / (double)real[i];
    }
    //printf("%lf %lf\n", accuracy / (double)len, are / len);
}

void experiment_filename(char* filename) {
    uint32 len;

    uint32* keys = read_keys(filename, &len);
    kernel_init<<<1, 1>>>();
    insert(keys, len);
    uint32* true_res = true_result(keys, &len);
    uint32* my_res = query(keys, len);
    compare(true_res, my_res, len);

    free(my_res);
    hipHostFree(keys);
    free(true_res);
}

int main() {
    char filename[100];
    printf("%d ", BATCH_SIZE);
    for (int i = 0; i < 10; ++i) {
        sprintf(filename, "../../data/%d.dat", i);
        experiment_filename(filename);
    }
    sort(expr_result, expr_result + 10);
    printf("%lf %lf %lf\n", 
        (expr_result[0] + expr_result[1]) / 2,
        (expr_result[4] + expr_result[5]) / 2,
        (expr_result[8] + expr_result[9]) / 2
    );
    return 0;
}